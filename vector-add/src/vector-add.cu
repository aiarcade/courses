#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"

const int N=10000000;





__global__
void add(int *A, int *B,int *R)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
    R[id] =  A[id]*B[id]*id;

}

int main()
{
	int *h_A,*h_B,*h_R;
	h_A=(int*)malloc(sizeof(int)*N);
	h_B=(int*)malloc(sizeof(int)*N);
	h_R=(int*)malloc(sizeof(int)*N);

	int i;
	for(i=0;i<N;i++){
			h_A[i]=i;
			h_B[i]=i;
			h_R[i]=88;
	}


	int *d_A,*d_B,*d_R;
	hipMalloc(&d_A,N*sizeof(int));
	hipMalloc(&d_B,N*sizeof(int));
	hipMalloc(&d_R,N*sizeof(int));


	hipMemcpy(d_A,h_A,N*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy(d_B,h_B,N*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy(d_R,h_R,N*sizeof(int), hipMemcpyHostToDevice );

	dim3 dimBlock( 512, 1, 1 );
	dim3 dimGrid( 1000, 1000 );

	add<<<dimGrid, dimBlock>>>(d_A, d_B,d_R);
	hipDeviceSynchronize();
	hipMemcpy(h_R,d_R,N*sizeof(int), hipMemcpyDeviceToHost );
	hipFree( d_R );
	hipFree( d_A );
	hipFree( d_A );

	for(i=0;i<10;i++)
		printf("%d\n",h_R[i]);
	free(h_A);
	free(h_B);
	free(h_R);

	return EXIT_SUCCESS;
}
